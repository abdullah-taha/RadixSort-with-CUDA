
#include <hip/hip_runtime.h>
#include <stdio.h>

//predicate function 
/*

test the first bit, returns an arrays of 1's and 0's called predicate
1 indicates that the first bit is zero, 0 indicates that the first bit is 1
and calculate the number of ones in the predicate and store it in dnumOfOnes

inputs :
d_in : input array that which predicate will be calculated (ex: 2, 3, 4, 6, 7, 1)
d_out : output array , result of predicate 			  	   (ex: 1, 0, 1, 1, 0, 0)
*/
__global__ void predicate(int* d_in, int* d_out, int* d_numOfOnes, int bitNo)
{
	if((d_in[threadIdx.x] & bitNo) == bitNo) {d_out[threadIdx.x] = 0;}
	else {d_out[threadIdx.x] = 1;atomicAdd(d_numOfOnes, 1);}
}


//flip_bits function
/*
flips all bit values in the input array (d_in) and store the result in the output array (d_out)
used in calculating the ters_predicate 
*/
__global__ void flip_bits(int *d_in,int *d_out)
{
	int indeks = threadIdx.x;
	d_out[indeks] = !d_in[indeks];
}


// scan function
/*
Blelloch scan : https://www.youtube.com/watch?v=mmYv3Haj6uc
every element in the output list is the sum of all the previous elements 
*/
__global__ void scan(int *d_in, int n)
{
	int indeks = threadIdx.x;
	int i;
	for(i=2; i <= n; i <<= 1)
	{
		
		if((indeks + 1) % i == 0)
		{
			//printf("inside if, indeks is %d \n",indeks);
			int offset = i >> 1;
			//printf("thread %d befor d_in[indeks] = %d offset is %d \n",indeks,d_in[indeks],offset);
			d_in[indeks] += d_in[indeks - offset];
			//printf("thread %d after d_in[indeks] = %d \n",indeks,d_in[indeks]);
		}
	}
	__syncthreads();

	// down sweep
	d_in[n-1] = 0;
	int j;
	for(j= i>>1; j>=2; j>>=1)
	{
		int offset = j >> 1;
		//printf("j is %d \n", j);
		if((indeks+1) % j == 0)
		{
			//printf("indeks is %d",indeks+1);
			int c = d_in[indeks];
			d_in[indeks] += d_in[indeks - offset];
			d_in[indeks - offset] = c; 
		}

	}
}


// sort function 
/*
here where the magic happens , we determine the new indexe for every element according to the followings:
for the i th element in the array, if the predicate is True(1), we move the element to the index in the i th element in the predicate scan array
if the predicate is False , we move the element to the index calculated by , indeks = corresonding value in the ters_predict_scan + numOfones

*/
__global__ void sort(int* d_input_array,int* d_output_array, int* d_predict, int* d_predict_scan, int* d_predict_numOfones, int* d_ters_predict, int* d_ters_predict_scan)
{
	int indeks = threadIdx.x;
	if(d_predict[indeks] == 1)
	{
		int new_indeks = d_predict_scan[indeks];
		d_output_array[new_indeks] = d_input_array[indeks];
	}

	else
	{
		int new_indeks = d_ters_predict_scan[indeks] + *d_predict_numOfones ;
		d_output_array[new_indeks] = d_input_array[indeks];
	}
} 

int main(void)
{


	// defining input array and fill it
	int *h_input_array = (int*)malloc(sizeof(int)*8);
	//for(int i=1;i<11;i++)h_input_array[i-1]=i;
	h_input_array[0]=7;
	h_input_array[1]=25;
	h_input_array[2]=2;
	h_input_array[3]=4;
	h_input_array[4]=70;
	h_input_array[5]=100;
	h_input_array[6]=8;
	h_input_array[7]=7;

	//print the input array
	//printf("array :\n");
	//for(int i=0;i<8;i++)printf("%d, ",h_input_array[i]);
	//printf("\n");

	//allocate memory on the host and device for the final sorted result array
	int* h_result_scan = (int*)malloc(sizeof(int)*8);
	int* d_result_scan;
	hipMalloc(&d_result_scan, sizeof(int)*8);

	// allocate memory on the host and device for the perdicate ters
	int* h_predicate_ters_result = (int*)malloc(sizeof(int)*8);
	int* d_predicate_ters_result;
	hipMalloc(&d_predicate_ters_result, sizeof(int)*8);

	//allocate memory on the host and device for the predicate result
	int* h_predicate_result = (int*)malloc(sizeof(int)*8);
	int *d_predicate_result;
	hipMalloc(&d_predicate_result, sizeof(int)*8);

	//allocate memory on the device for the input array
	int* d_input_array;
	hipMalloc(&d_input_array, sizeof(int)*8);
	hipMemcpy(d_input_array, h_input_array,sizeof(int)*8, hipMemcpyHostToDevice);

	//allocate memory on the device for the number of ones in the predicate result 
	int* d_numOfOnes;
	int* h_numOfOnes = (int*)malloc(sizeof(int));
	hipMalloc(&d_numOfOnes, sizeof(int));

	//allocate memory on the host for the scan result array
	int* h_ters_predict_scan = (int*)malloc(sizeof(int)*8);
	int* d_result_ters_scan;
	hipMalloc(&d_result_ters_scan, sizeof(int)*8);

	//allocate memory on host and device for output sorted array
	int* h_sort_result = (int*)malloc(sizeof(int)*8);
	int* d_sort_result;
	hipMalloc(&d_sort_result, sizeof(int)*8);

// bitmap is a mask to be used in bitwise operations , initial value is 1 to test the first bit
int bitmap = 1;
for(int k=0;k<32;k++)
{
	//print array at every step to watch the sorting
	printf("array :\n");
	for(int i=0;i<8;i++)printf("%d, ",h_input_array[i]);
	printf("\n");

	//set the numOfOnes to 0 at every iteration
	hipMemset(d_numOfOnes,0,sizeof(int));

	// call the predicate kernel 
	predicate<<<1,8>>>(d_input_array,d_predicate_result,d_numOfOnes,bitmap);

	//copy the predicate result and number of ones from the device to  the host
	hipMemcpy(h_predicate_result, d_predicate_result,sizeof(int)*8, hipMemcpyDeviceToHost);
	hipMemcpy(h_numOfOnes, d_numOfOnes,sizeof(int), hipMemcpyDeviceToHost);

	// print the predicate array and number of ones
	printf("predicate :\n");
	for(int i=0;i<8;i++)printf(" %d, ",h_predicate_result[i]);
	printf("\n");
	printf("num of ones : %d \n",*h_numOfOnes);

	//copy the predicate result from host to the device and store it in d_result scan. the change will be applied on the same array
	hipMemcpy(d_result_scan, h_predicate_result,sizeof(int)*8, hipMemcpyHostToDevice);

	//invoke the kernal function
	scan<<<1,8>>>(d_result_scan,8);

	//copy the result back to the host 
	hipMemcpy(h_result_scan, d_result_scan,sizeof(int)*8, hipMemcpyDeviceToHost);

	//print the result
	printf("predicate scan result :\n");
	for(int i=0;i<8;i++)printf(" %d, ",h_result_scan[i]);
	printf("\n");

	//call the flip bits kernel on the device
	flip_bits<<<1,8>>>(d_predicate_result,d_predicate_ters_result);

	//copy the result to the host
	hipMemcpy(h_predicate_ters_result,d_predicate_ters_result,sizeof(int)*8,hipMemcpyDeviceToHost);

	//print the result
	printf("predict ters :\n");
	for(int i=0;i<8;i++)printf(" %d, ",h_predicate_ters_result[i]);
	printf("\n");

	//copy the !predicate from the host to the device and store it in d_result_ters_scan
	hipMemcpy(d_result_ters_scan, h_predicate_ters_result,sizeof(int)*8, hipMemcpyHostToDevice);

	//call the scan upon d_result_ters_scan, the change will be applied ont he same array
	scan<<<1,8>>>(d_result_ters_scan,8);

	//copy the result to the host h_ters_predicate_scan
	hipMemcpy(h_ters_predict_scan, d_result_ters_scan,sizeof(int)*8, hipMemcpyDeviceToHost);

	//print the result
	printf("ters predicate scan result :\n");
	for(int i=0;i<8;i++)printf(" %d, ",h_ters_predict_scan[i]);
	printf("\n");


	//invoke the sorting function on the kernel
	sort<<<1,8>>>(d_input_array, d_sort_result, d_predicate_result, d_result_scan, d_numOfOnes, d_predicate_ters_result, d_result_ters_scan );
	//copy the sorted list back to the host and print it 
	hipMemcpy(h_sort_result, d_sort_result,sizeof(int)*8, hipMemcpyDeviceToHost);
	printf("\n\n\n\n\nSORTED WITH CUDA !!!!!!!!!!:\n");
	for(int i=0;i<8;i++)printf(" %d, ",h_sort_result[i]);

	//printf("bitmap %d ",bitmap);

	//update the mask to test the next bit 
	bitmap <<= 1;

	//update the input array for every iteration
	memcpy(h_input_array,h_sort_result, 8 * sizeof(int));
	//update the input array on the device
	hipMemcpy(d_input_array, h_input_array,sizeof(int)*8, hipMemcpyHostToDevice);
}

  	hipFree(d_input_array);
  	hipFree(d_sort_result);
  	hipFree(d_result_scan);
  	hipFree(d_predicate_result);
  	hipFree(d_numOfOnes);
  	hipFree(d_predicate_ters_result);
  	hipFree(d_result_ters_scan);
	return 0;

	
}